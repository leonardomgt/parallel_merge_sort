#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "mpi.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>


void merge(int *src, ulong offset, ulong leftSize, ulong rightSize);
void sequentialMergeSort(int *src, ulong offset, ulong size);
void parallelMergeSort(int *src, ulong size, int *result, bool gpu_mode);
__global__ void gpuMergeSort(int *src, int *dest, ulong size, ulong threadPartSize, int nSlices, int threadsPerBlock, int blocksPerGrid);
__device__ void gpuMerge(int* src, int* dest, ulong it_left, ulong it_middle, ulong it_right);
// * Create new MPI datatype: MPI_EDGE
MPI_Datatype create_chunk_MPI_datatype();

struct chunk {
	int* data;
	ulong size;
};

int main(int argc, char **argv)
{

	// * Check arguments
	if (argc != 4)
	{
		fprintf(stderr, "usage: %s <random_seed> <dataset_size> <execution_mode:'GPU/CPU'>\n", argv[0]);
		exit(EXIT_FAILURE);
	}

	int base_seed;
	ulong ds_size;

	sscanf(argv[1], "%010x", &base_seed);
	sscanf(argv[2], "%lu", &ds_size);

	bool gpu_mode = !strcmp(argv[3], "GPU");

	// * Initalize MPI Library
	MPI_Init(&argc, &argv);

	int n_procs, proc_id;
	MPI_Comm_size(MPI_COMM_WORLD, &n_procs);
	MPI_Comm_rank(MPI_COMM_WORLD, &proc_id);

	//printf("ID: %d\n", proc_id);
	int *my_array = new int[ds_size];

	if (proc_id == 0)
	{

		printf("Parallel Merge Sort\n\n");
		printf("Random seed: %s\n", argv[1]);
		printf("Data size: %s\n", argv[2]);
		printf("Execution mode: %s\n\n", gpu_mode ? "CPU + GPU" : "CPU");
		printf("Number of CPU nodes: %d\n", n_procs);
		

		double start_generating = MPI_Wtime();

		// * Generate differente seed for each node, such that different numbers are generated.
		srand(base_seed + proc_id);

		// * Generate N pseudo-random integers in the interval [0, RAND_MAX]
		for (size_t i = 0; i < ds_size; i++)
			my_array[i] = rand();

		double end_generating = MPI_Wtime();

		printf("Generating complete in %f s\n", end_generating - start_generating);
	}

	double start_exec = MPI_Wtime();

	// * Execute the Parallel Merge Sort algorithm.

	int *merged_array = new int[ds_size];
	parallelMergeSort(my_array, ds_size, merged_array, gpu_mode);

	double end_exec = MPI_Wtime();


	if (proc_id == 0)
	{
		printf("Execution complete in %f s\n", end_exec - start_exec);

		printf("\n\nSorted array...\n\n");

		for (ulong i = 0; i < ds_size; i += (ds_size / 20 + 1))
		{
			printf("i: %lu, v: %d\n", i, merged_array[i]);
		}
	}

	MPI_Finalize();
	return 0;
}

void parallelMergeSort(int *src, ulong size, int *result, bool gpu_mode)
{

	int n_procs, proc_id;
	MPI_Comm_size(MPI_COMM_WORLD, &n_procs);
	MPI_Comm_rank(MPI_COMM_WORLD, &proc_id);

	// * Evenly number of values to be sorted per process
	ulong valuesPerProcess = ((size + n_procs - 1) / n_procs);
	
	int *processPart = new int[valuesPerProcess];

	// TODO: Use MPI_Scatterv to distribute the values more evenly

	MPI_Scatter(
		src,
		valuesPerProcess, MPI_INT,
		processPart,
		valuesPerProcess, MPI_INT,
		0, MPI_COMM_WORLD);

	// * Adjust valuesPerProcess for the last process.
	if (proc_id == n_procs - 1 && size % valuesPerProcess != 0)
	{
		valuesPerProcess = size % valuesPerProcess;
	}


	if(gpu_mode){

		// * Calculate threadsPerBlock and blocksPerGrid
		int threadsPerBlock;  
		int minBlocksPerGrid; 
						
		int blocksPerGrid;

		hipOccupancyMaxPotentialBlockSize(&minBlocksPerGrid, &threadsPerBlock,	gpuMergeSort);

		blocksPerGrid = (valuesPerProcess + threadsPerBlock - 1) / threadsPerBlock;

		// * Allocate and copy data to GPU shared memory
		int *gpuData, *gpuResult;


		checkCudaErrors(hipMalloc((void**) &gpuData, valuesPerProcess * sizeof(int)));
		checkCudaErrors(hipMalloc((void**) &gpuResult, valuesPerProcess * sizeof(int)));

		checkCudaErrors(hipMemcpy(gpuData, processPart, valuesPerProcess * sizeof(int), hipMemcpyHostToDevice));

		int *gpuInput = gpuData, *gpuOutput = gpuResult;

		long nThreads = threadsPerBlock * blocksPerGrid;

		for (int valuesPerThread = 2; valuesPerThread < (valuesPerProcess << 1); valuesPerThread <<= 1) {
			ulong nSlices = valuesPerProcess / ((nThreads) * valuesPerThread) + 1;
	
			gpuMergeSort<<<blocksPerGrid, threadsPerBlock>>>(gpuInput, gpuOutput, valuesPerProcess, valuesPerThread, nSlices, threadsPerBlock, blocksPerGrid);
	
	
			// Swap input and output pointers for next iteration
			gpuInput = gpuInput == gpuData ? gpuResult : gpuData;
			gpuOutput = gpuOutput == gpuData ? gpuResult : gpuData;
		}
		
		checkCudaErrors(hipMemcpy(result, gpuData, valuesPerProcess * sizeof(int), hipMemcpyDeviceToHost));

		checkCudaErrors(hipFree(gpuInput));
		checkCudaErrors(hipFree(gpuOutput));
		// gpuMergeSort<<<blocksPerGrid, threadsPerBlock>>>(processPart, nThreads, valuesPerProcess);
	}
	else{
		sequentialMergeSort(processPart, 0, valuesPerProcess);
		memcpy(result, processPart, valuesPerProcess * sizeof(int));
	}


	int countToSend = valuesPerProcess;

	MPI_Status status;

	for (size_t k = 1; k <= ceil(log2(n_procs)); k++)
	{
		if (proc_id % (int)pow(2, k) == pow(2, k - 1))
		{
			// printf("\nId %d sending %d values to %f\n", proc_id, countToSend, proc_id - pow(2, k - 1));
			MPI_Send(result, countToSend, MPI_INT, proc_id - pow(2, k - 1), k, MPI_COMM_WORLD);
		}
		else if (proc_id % (int)pow(2, k) == 0 && proc_id + pow(2, k - 1) < n_procs)
		{
			// printf("\nId %d receiving %d values from %f\n", proc_id, valuesPerProcess * (int)pow(2, k), proc_id + pow(2, k - 1));
			MPI_Recv(result + countToSend, valuesPerProcess * (int)pow(2, k), MPI_INT, proc_id + pow(2, k - 1), k, MPI_COMM_WORLD, &status);

			int receivedCount;
			MPI_Get_count(&status, MPI_INT, &receivedCount);

			merge(result, 0, countToSend, receivedCount);

			countToSend += receivedCount;
		}
	}
}

void merge(int *src, ulong offset, ulong leftSize, ulong rightSize)
{

	int *left = new int[leftSize];
	int *right = new int[rightSize];

	memcpy(left, src + offset, leftSize * sizeof(int));
	memcpy(right, src + offset + leftSize, rightSize * sizeof(int));

	int it_left = 0, it_right = 0, it_res = offset;

	while (it_left < leftSize && it_right < rightSize)
	{
		if (left[it_left] < right[it_right])
		{
			src[it_res] = left[it_left];
			it_left++;
		}
		else
		{
			src[it_res] = right[it_right];
			it_right++;
		}
		it_res++;
	}

	while (it_left < leftSize)
	{
		src[it_res] = left[it_left];
		it_left++;
		it_res++;
	}

	while (it_right < rightSize)
	{
		src[it_res] = right[it_right];
		it_right++;
		it_res++;
	}
}

void sequentialMergeSort(int *src, ulong offset, ulong size)
{
	if (size > 1)
	{
		int leftSize = size / 2;
		int rightSize = size - leftSize;

		sequentialMergeSort(src, offset, leftSize);
		sequentialMergeSort(src, offset + leftSize, rightSize);

		merge(src, offset, leftSize, rightSize);
	}
}

__global__
void gpuMergeSort(int *src, int *dest, ulong size, ulong threadPartSize, int nSlices, int threadsPerBlock, int blocksPerGrid)
{
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

	ulong it_left = threadPartSize * threadIndex * nSlices;
	ulong it_middle, it_right;

    for (long si = 0; si < nSlices; si++) {
		if (it_left >= size)
			break;

		it_middle = min(it_left + (threadPartSize >> 1), size);
		it_right = min(it_left + threadPartSize, size);

		gpuMerge(src, dest, it_left, it_middle, it_right);
		it_left += threadPartSize;
	}
}

__device__ 
void gpuMerge(int* src, int* dest, ulong it_left, ulong it_middle, ulong it_right) {
    ulong i = it_left;
    ulong j = it_middle;
    for (ulong k = it_left; k < it_right; k++) {
        if (i < it_middle && (j >= it_right || src[i] < src[j])) {
            dest[k] = src[i];
            i++;
        } else {
            dest[k] = src[j];
            j++;
        }
    }
}