#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "mpi.h"

#include "hip/hip_runtime.h"
#include ""


void merge(int *src, int offset, int leftSize, int rightSize);
void sequentialMergeSort(int *src, int offset, int size);
void parallelMergeSort(int *src, int size, int *result, bool gpu_mode);
void gpuMergeSort(int *src, int offset, int size);

int main(int argc, char **argv)
{

	// * Check arguments
	if (argc != 4)
	{
		fprintf(stderr, "usage: %s <random_seed> <dataset_size> <execution_mode:'GPU/CPU'>\n", argv[0]);
		exit(EXIT_FAILURE);
	}

	int base_seed;
	ulong ds_size;

	sscanf(argv[1], "%010x", &base_seed);
	sscanf(argv[2], "%lu", &ds_size);

	bool gpu_mode = !strcmp(argv[3], "GPU");

	// * Initalize MPI Library
	MPI_Init(&argc, &argv);

	int n_procs, proc_id;
	MPI_Comm_size(MPI_COMM_WORLD, &n_procs);
	MPI_Comm_rank(MPI_COMM_WORLD, &proc_id);

	//printf("ID: %d\n", proc_id);

	int *my_array = new int[ds_size];

	if (proc_id == 0)
	{

		printf("Parallel Merge Sort\n\n");
		printf("Random seed: %s\n", argv[1]);
		printf("Data size: %s\n", argv[2]);
		printf("Execution mode: %s\n\n", gpu_mode ? "CPU + GPU" : "CPU");
		printf("Number of CPU nodes: %d\n", n_procs);
		

		double start_generating = MPI_Wtime();

		// * Generate differente seed for each node, such that different numbers are generated.
		srand(base_seed + proc_id);

		// * Generate N pseudo-random integers in the interval [0, RAND_MAX]
		for (size_t i = 0; i < ds_size; i++)
			my_array[i] = rand();

		double end_generating = MPI_Wtime();

		printf("Generating complete in %f s\n", end_generating - start_generating);
	}

	double start_exec = MPI_Wtime();

	// * Execute the Parallel Merge Sort algorithm.

	int *merged_array = new int[ds_size];
	parallelMergeSort(my_array, ds_size, merged_array, gpu_mode);

	double end_exec = MPI_Wtime();

	if (proc_id == 0)
	{
		printf("Execution complete in %f s\n", end_exec - start_exec);

		printf("\n\nSorted array...\n\n");

		for (ulong i = 0; i < ds_size; i += (ds_size / 20 + 1))
		{
			printf("i: %lu, v: %d\n", i, merged_array[i]);
		}
	}

	MPI_Finalize();
	return 0;
}

void parallelMergeSort(int *src, int size, int *result, bool gpu_mode)
{

	int n_procs, proc_id;
	MPI_Comm_size(MPI_COMM_WORLD, &n_procs);
	MPI_Comm_rank(MPI_COMM_WORLD, &proc_id);

	// * Evenly number of values to be sorted per process
	int valuesPerProcess = (size + n_procs - 1) / n_procs;

	int *processPart = new int[valuesPerProcess];

	// TODO: Use MPI_Scatterv to distribute the values more evenly

	MPI_Scatter(
		src,
		valuesPerProcess, MPI_INT,
		processPart,
		valuesPerProcess, MPI_INT,
		0, MPI_COMM_WORLD);

	// * Adjust valuesPerProcess for the last process.
	if (proc_id == n_procs - 1 && size % valuesPerProcess != 0)
	{
		valuesPerProcess = size % valuesPerProcess;
	}


	if(gpu_mode){
		// TODO: GPU Parallelization 
		gpuMergeSort(processPart, 0, valuesPerProcess);
	}
	else{
		sequentialMergeSort(processPart, 0, valuesPerProcess);
	}

	memcpy(result, processPart, valuesPerProcess * sizeof(int));

	int countToSend = valuesPerProcess;

	MPI_Status status;

	for (size_t k = 1; k <= ceil(log2(n_procs)); k++)
	{
		if (proc_id % (int)pow(2, k) == pow(2, k - 1))
		{
			// printf("\nId %d sending to %f\n", proc_id, proc_id - pow(2, k - 1));
			MPI_Send(result, countToSend, MPI_INT, proc_id - pow(2, k - 1), k, MPI_COMM_WORLD);
		}
		else if (proc_id % (int)pow(2, k) == 0 && proc_id + pow(2, k - 1) < n_procs)
		{
			// printf("\nId %d receiving from %f\n", proc_id, proc_id + pow(2, k - 1));
			MPI_Recv(result + countToSend, valuesPerProcess * k, MPI_INT, proc_id + pow(2, k - 1), k, MPI_COMM_WORLD, &status);

			int receivedCount;
			MPI_Get_count(&status, MPI_INT, &receivedCount);

			merge(result, 0, countToSend, receivedCount);

			countToSend += receivedCount;
		}
	}
}

void merge(int *src, int offset, int leftSize, int rightSize)
{

	int *left = new int[leftSize];
	int *right = new int[rightSize];

	memcpy(left, src + offset, leftSize * sizeof(int));
	memcpy(right, src + offset + leftSize, rightSize * sizeof(int));

	int it_left = 0, it_right = 0, it_res = offset;

	while (it_left < leftSize && it_right < rightSize)
	{
		if (left[it_left] < right[it_right])
		{
			src[it_res] = left[it_left];
			it_left++;
		}
		else
		{
			src[it_res] = right[it_right];
			it_right++;
		}
		it_res++;
	}

	while (it_left < leftSize)
	{
		src[it_res] = left[it_left];
		it_left++;
		it_res++;
	}

	while (it_right < rightSize)
	{
		src[it_res] = right[it_right];
		it_right++;
		it_res++;
	}
}

void sequentialMergeSort(int *src, int offset, int size)
{
	if (size > 1)
	{
		int leftSize = size / 2;
		int rightSize = size - leftSize;

		sequentialMergeSort(src, offset, leftSize);
		sequentialMergeSort(src, offset + leftSize, rightSize);

		merge(src, offset, leftSize, rightSize);
	}
}

void gpuMergeSort(int *src, int offset, int size)
{
	hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Device Number: %d\n", 0);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	
}
